#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// Squared_Difference_GPU.cu
//
// Code generation for function 'Squared_Difference_GPU'
//

// Include files
#include "Squared_Difference_GPU.h"
#include "Squared_Difference_GPU_data.h"
#include "Squared_Difference_GPU_emxutil.h"
#include "Squared_Difference_GPU_types.h"
#include "rt_nonfinite.h"
#include "rtwhalf.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWMemoryManager.hpp"
#include <cstdio>
#include <cstdlib>
#include <cstring>

// Type Definitions
struct emxArray___half {
  __half *data;
  int32_T *size;
  int32_T allocatedSize;
  int32_T numDimensions;
  boolean_T canFreeData;
};

// Function Declarations
static __global__ void
Squared_Difference_GPU_kernel1(const emxArray_real32_T true_database,
                               const int32_T b_true_database,
                               emxArray___half c_true_database);

static __global__ void
Squared_Difference_GPU_kernel2(const emxArray_real_T Meas_Cts_temp,
                               const int32_T b_Meas_Cts_temp,
                               emxArray___half c_Meas_Cts_temp);

static __global__ void
Squared_Difference_GPU_kernel3(const int32_T true_database,
                               emxArray___half Squ_diff);

static __global__ void Squared_Difference_GPU_kernel4(
    const emxArray___half true_database, const emxArray___half Meas_Cts_temp,
    const int32_T b_true_database, const int32_T i, const int32_T Squ_diff_dim0,
    const int32_T true_database_dim0, emxArray___half Squ_diff);

static void checkCudaError(hipError_t errCode, const char_T *file,
                           uint32_T line);

static void gpuEmxEnsureCapacity_real16_T(const emxArray_real16_T *cpu,
                                          emxArray___half *gpu);

static void gpuEmxFree_real16_T(emxArray___half *gpu);

static void gpuEmxFree_real32_T(emxArray_real32_T *gpu);

static void gpuEmxFree_real_T(emxArray_real_T *gpu);

static void gpuEmxMemcpyCpuToGpu_real16_T(emxArray___half *gpu,
                                          const emxArray_real16_T *cpu);

static void gpuEmxMemcpyCpuToGpu_real32_T(emxArray_real32_T *gpu,
                                          const emxArray_real32_T *cpu);

static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu);

static void gpuEmxMemcpyGpuToCpu_real16_T(emxArray_real16_T *cpu,
                                          emxArray___half *gpu);

static void gpuEmxReset_real16_T(emxArray___half *gpu);

static void gpuEmxReset_real32_T(emxArray_real32_T *gpu);

static void gpuEmxReset_real_T(emxArray_real_T *gpu);

static void raiseCudaError(uint32_T errCode, const char_T *file, uint32_T line,
                           const char_T *errorName, const char_T *errorString);

// Function Definitions
static __global__
    __launch_bounds__(1024, 1) void Squared_Difference_GPU_kernel1(
        const emxArray_real32_T true_database, const int32_T b_true_database,
        emxArray___half c_true_database)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_true_database);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    c_true_database.data[i] = static_cast<__half>(true_database.data[i]);
  }
}

static __global__
    __launch_bounds__(1024, 1) void Squared_Difference_GPU_kernel2(
        const emxArray_real_T Meas_Cts_temp, const int32_T b_Meas_Cts_temp,
        emxArray___half c_Meas_Cts_temp)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_Meas_Cts_temp);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    c_Meas_Cts_temp.data[i] = static_cast<__half>(Meas_Cts_temp.data[i]);
  }
}

static __global__ __launch_bounds__(
    1024, 1) void Squared_Difference_GPU_kernel3(const int32_T true_database,
                                                 emxArray___half Squ_diff)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(true_database);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    Squ_diff.data[i] = __float2half(0.0F);
  }
}

static __global__
    __launch_bounds__(1024, 1) void Squared_Difference_GPU_kernel4(
        const emxArray___half true_database,
        const emxArray___half Meas_Cts_temp, const int32_T b_true_database,
        const int32_T i, const int32_T Squ_diff_dim0,
        const int32_T true_database_dim0, emxArray___half Squ_diff)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b_true_database) + 1ULL) *
                (static_cast<uint64_T>(i - 1) + 1ULL) -
            1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T b_i;
    int32_T j;
    __half h;
    j = static_cast<int32_T>(idx %
                             (static_cast<uint64_T>(b_true_database) + 1ULL));
    b_i = static_cast<int32_T>((idx - static_cast<uint64_T>(j)) /
                               (static_cast<uint64_T>(b_true_database) + 1ULL));
    h = Meas_Cts_temp.data[j];
    Squ_diff.data[b_i + Squ_diff_dim0 * j] =
        (true_database.data[b_i + true_database_dim0 * j] - h) *
        (true_database.data[b_i + true_database_dim0 * j] - h);
  }
}

static void checkCudaError(hipError_t errCode, const char_T *file,
                           uint32_T line)
{
  if (errCode != hipSuccess) {
    raiseCudaError(errCode, file, line, hipGetErrorName(errCode),
                   hipGetErrorString(errCode));
  }
}

static void gpuEmxEnsureCapacity_real16_T(const emxArray_real16_T *cpu,
                                          emxArray___half *gpu)
{
  __half *newData;
#define CUDACHECK(errCall) checkCudaError(errCall, __FILE__, __LINE__)
  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  if (gpu->data == 0) {
    newData = 0ULL;
    mwCudaMalloc(&newData,
                 static_cast<uint64_T>(cpu->allocatedSize * sizeof(__half)));
    CUDACHECK(hipGetLastError());
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = (int32_T *)emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T));
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      gpu->size[i] = cpu->size[i];
    }
    gpu->allocatedSize = cpu->allocatedSize;
    gpu->canFreeData = true;
    gpu->data = newData;
  } else {
    int32_T actualSizeCpu;
    int32_T actualSizeGpu;
    actualSizeCpu = 1;
    actualSizeGpu = 1;
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      actualSizeGpu *= gpu->size[i];
      actualSizeCpu *= cpu->size[i];
      gpu->size[i] = cpu->size[i];
    }
    if (gpu->allocatedSize < actualSizeCpu) {
      newData = 0ULL;
      mwCudaMalloc(&newData,
                   static_cast<uint64_T>(cpu->allocatedSize * sizeof(__half)));
      CUDACHECK(hipGetLastError());
      hipMemcpy(newData, gpu->data, actualSizeGpu * sizeof(__half),
                 hipMemcpyDeviceToDevice);
      CUDACHECK(hipGetLastError());
      gpu->allocatedSize = cpu->allocatedSize;
      if (gpu->canFreeData) {
        mwCudaFree(gpu->data);
        CUDACHECK(hipGetLastError());
      }
      gpu->canFreeData = true;
      gpu->data = newData;
    }
  }
#undef CUDACHECK
}

static void gpuEmxFree_real16_T(emxArray___half *gpu)
{
#define CUDACHECK(errCall) checkCudaError(errCall, __FILE__, __LINE__)
  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  if (gpu->data != (void *)4207599121ULL) {
    mwCudaFree(gpu->data);
    CUDACHECK(hipGetLastError());
  }
  emlrtFreeMex(gpu->size);
#undef CUDACHECK
}

static void gpuEmxFree_real32_T(emxArray_real32_T *gpu)
{
#define CUDACHECK(errCall) checkCudaError(errCall, __FILE__, __LINE__)
  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  if (gpu->data != (void *)4207599121ULL) {
    mwCudaFree(gpu->data);
    CUDACHECK(hipGetLastError());
  }
  emlrtFreeMex(gpu->size);
#undef CUDACHECK
}

static void gpuEmxFree_real_T(emxArray_real_T *gpu)
{
#define CUDACHECK(errCall) checkCudaError(errCall, __FILE__, __LINE__)
  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  if (gpu->data != (void *)4207599121ULL) {
    mwCudaFree(gpu->data);
    CUDACHECK(hipGetLastError());
  }
  emlrtFreeMex(gpu->size);
#undef CUDACHECK
}

static void gpuEmxMemcpyCpuToGpu_real16_T(emxArray___half *gpu,
                                          const emxArray_real16_T *cpu)
{
  int32_T actualSize;
  int32_T i;
#define CUDACHECK(errCall) checkCudaError(errCall, __FILE__, __LINE__)
  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  if (gpu->numDimensions < cpu->numDimensions) {
    gpu->numDimensions = cpu->numDimensions;
    emlrtFreeMex(gpu->size);
    gpu->size = (int32_T *)emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T));
  } else {
    gpu->numDimensions = cpu->numDimensions;
  }
  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < actualSize) {
    if (gpu->canFreeData) {
      mwCudaFree(gpu->data);
      CUDACHECK(hipGetLastError());
    }
    i = cpu->allocatedSize;
    if (i < actualSize) {
      i = actualSize;
    }
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
    mwCudaMalloc(&gpu->data,
                 static_cast<uint64_T>(gpu->allocatedSize * sizeof(real16_T)));
    CUDACHECK(hipGetLastError());
  }
  hipMemcpy(gpu->data, cpu->data, actualSize * sizeof(real16_T),
             hipMemcpyHostToDevice);
  CUDACHECK(hipGetLastError());
#undef CUDACHECK
}

static void gpuEmxMemcpyCpuToGpu_real32_T(emxArray_real32_T *gpu,
                                          const emxArray_real32_T *cpu)
{
  int32_T actualSize;
  int32_T i;
#define CUDACHECK(errCall) checkCudaError(errCall, __FILE__, __LINE__)
  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  if (gpu->numDimensions < cpu->numDimensions) {
    gpu->numDimensions = cpu->numDimensions;
    emlrtFreeMex(gpu->size);
    gpu->size = (int32_T *)emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T));
  } else {
    gpu->numDimensions = cpu->numDimensions;
  }
  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < actualSize) {
    if (gpu->canFreeData) {
      mwCudaFree(gpu->data);
      CUDACHECK(hipGetLastError());
    }
    i = cpu->allocatedSize;
    if (i < actualSize) {
      i = actualSize;
    }
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
    mwCudaMalloc(&gpu->data,
                 static_cast<uint64_T>(gpu->allocatedSize * sizeof(real32_T)));
    CUDACHECK(hipGetLastError());
  }
  hipMemcpy(gpu->data, cpu->data, actualSize * sizeof(real32_T),
             hipMemcpyHostToDevice);
  CUDACHECK(hipGetLastError());
#undef CUDACHECK
}

static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu)
{
  int32_T actualSize;
  int32_T i;
#define CUDACHECK(errCall) checkCudaError(errCall, __FILE__, __LINE__)
  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  if (gpu->numDimensions < cpu->numDimensions) {
    gpu->numDimensions = cpu->numDimensions;
    emlrtFreeMex(gpu->size);
    gpu->size = (int32_T *)emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T));
  } else {
    gpu->numDimensions = cpu->numDimensions;
  }
  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < actualSize) {
    if (gpu->canFreeData) {
      mwCudaFree(gpu->data);
      CUDACHECK(hipGetLastError());
    }
    i = cpu->allocatedSize;
    if (i < actualSize) {
      i = actualSize;
    }
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
    mwCudaMalloc(&gpu->data,
                 static_cast<uint64_T>(gpu->allocatedSize * sizeof(real_T)));
    CUDACHECK(hipGetLastError());
  }
  hipMemcpy(gpu->data, cpu->data, actualSize * sizeof(real_T),
             hipMemcpyHostToDevice);
  CUDACHECK(hipGetLastError());
#undef CUDACHECK
}

static void gpuEmxMemcpyGpuToCpu_real16_T(emxArray_real16_T *cpu,
                                          emxArray___half *gpu)
{
  int32_T actualSize;
#define CUDACHECK(errCall) checkCudaError(errCall, __FILE__, __LINE__)
  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  hipMemcpy(cpu->data, gpu->data, actualSize * sizeof(real16_T),
             hipMemcpyDeviceToHost);
  CUDACHECK(hipGetLastError());
#undef CUDACHECK
}

static void gpuEmxReset_real16_T(emxArray___half *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_real16_T));
}

static void gpuEmxReset_real32_T(emxArray_real32_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_real32_T));
}

static void gpuEmxReset_real_T(emxArray_real_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_real_T));
}

static void raiseCudaError(uint32_T errCode, const char_T *file, uint32_T line,
                           const char_T *errorName, const char_T *errorString)
{
  emlrtRTEInfo rtInfo;
  uint32_T len;
  char_T *brk;
  char_T *fn;
  char_T *pn;
  len = strlen(file);
  pn = (char_T *)calloc(len + 1U, 1U);
  fn = (char_T *)calloc(len + 1U, 1U);
  memcpy(pn, file, len);
  memcpy(fn, file, len);
  brk = strrchr(fn, '.');
  *brk = '\x00';
  brk = strrchr(fn, '/');
  if (brk == nullptr) {
    brk = strrchr(fn, '\\');
  }
  if (brk == nullptr) {
    brk = fn;
  } else {
    brk++;
  }
  rtInfo.lineNo = static_cast<int32_T>(line);
  rtInfo.colNo = 0;
  rtInfo.fName = brk;
  rtInfo.pName = pn;
  emlrtCUDAError(errCode, (char_T *)errorName, (char_T *)errorString, &rtInfo,
                 emlrtRootTLSGlobal);
}

void Squared_Difference_GPU(const emxArray_real32_T *true_database,
                            const emxArray_real_T *Meas_Cts_temp,
                            emxArray_real16_T *Squ_diff)
{
  dim3 block;
  dim3 grid;
  emxArray___half b_gpu_Meas_Cts_temp;
  emxArray___half b_gpu_true_database;
  emxArray___half gpu_Squ_diff;
  emxArray_real16_T *b_Meas_Cts_temp;
  emxArray_real16_T *b_true_database;
  emxArray_real32_T gpu_true_database;
  emxArray_real_T gpu_Meas_Cts_temp;
  int32_T Squ_diff_dim0;
  int32_T c_true_database;
  int32_T i;
  int32_T true_database_dim0;
  boolean_T Squ_diff_dirtyOnCpu;
  boolean_T Squ_diff_dirtyOnGpu;
  boolean_T validLaunchParams;
#define CUDACHECK(errCall) checkCudaError(errCall, __FILE__, __LINE__)
  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  gpuEmxReset_real16_T(&gpu_Squ_diff);
  gpuEmxReset_real16_T(&b_gpu_Meas_Cts_temp);
  gpuEmxReset_real_T(&gpu_Meas_Cts_temp);
  gpuEmxReset_real16_T(&b_gpu_true_database);
  gpuEmxReset_real32_T(&gpu_true_database);
  Squ_diff_dirtyOnGpu = false;
  Squ_diff_dirtyOnCpu = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_real16_T(&b_true_database, 2, true);
  // true_database=single(ones(100000000,20));
  // Meas_Cts_temp=double(ones(1,20));
  // tic;
  i = b_true_database->size[0] * b_true_database->size[1];
  b_true_database->size[0] = true_database->size[0];
  b_true_database->size[1] = true_database->size[1];
  emxEnsureCapacity_real16_T(b_true_database, i);
  gpuEmxEnsureCapacity_real16_T(b_true_database, &b_gpu_true_database);
  c_true_database = true_database->size[0] * true_database->size[1] - 1;
  validLaunchParams = mwGetLaunchParameters1D(
      static_cast<real_T>(c_true_database + 1LL), &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    gpuEmxMemcpyCpuToGpu_real32_T(&gpu_true_database, true_database);
    Squared_Difference_GPU_kernel1<<<grid, block>>>(
        gpu_true_database, c_true_database, b_gpu_true_database);
    CUDACHECK(hipGetLastError());
  }
  emxInit_real16_T(&b_Meas_Cts_temp, 2, true);
  i = b_Meas_Cts_temp->size[0] * b_Meas_Cts_temp->size[1];
  b_Meas_Cts_temp->size[0] = 1;
  b_Meas_Cts_temp->size[1] = Meas_Cts_temp->size[1];
  emxEnsureCapacity_real16_T(b_Meas_Cts_temp, i);
  gpuEmxEnsureCapacity_real16_T(b_Meas_Cts_temp, &b_gpu_Meas_Cts_temp);
  i = Meas_Cts_temp->size[1] - 1;
  validLaunchParams = mwGetLaunchParameters1D(static_cast<real_T>(i + 1LL),
                                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    gpuEmxMemcpyCpuToGpu_real_T(&gpu_Meas_Cts_temp, Meas_Cts_temp);
    Squared_Difference_GPU_kernel2<<<grid, block>>>(gpu_Meas_Cts_temp, i,
                                                    b_gpu_Meas_Cts_temp);
    CUDACHECK(hipGetLastError());
  }
  // toc;
  i = Squ_diff->size[0] * Squ_diff->size[1];
  Squ_diff->size[0] = b_true_database->size[0];
  Squ_diff->size[1] = b_true_database->size[1];
  emxEnsureCapacity_real16_T(Squ_diff, i);
  c_true_database = b_true_database->size[0] * b_true_database->size[1] - 1;
  validLaunchParams = mwGetLaunchParameters1D(
      static_cast<real_T>(c_true_database + 1LL), &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    gpuEmxMemcpyCpuToGpu_real16_T(&gpu_Squ_diff, Squ_diff);
    Squared_Difference_GPU_kernel3<<<grid, block>>>(c_true_database,
                                                    gpu_Squ_diff);
    CUDACHECK(hipGetLastError());
    Squ_diff_dirtyOnCpu = false;
    Squ_diff_dirtyOnGpu = true;
  }
  i = b_true_database->size[0];
  c_true_database = b_true_database->size[1] - 1;
  Squ_diff_dim0 = Squ_diff->size[0];
  true_database_dim0 = b_true_database->size[0];
  validLaunchParams = mwGetLaunchParameters1D(
      static_cast<real_T>((c_true_database + 1LL) * ((i - 1) + 1LL)), &grid,
      &block, 1024U, 65535U);
  if (validLaunchParams) {
    if (Squ_diff_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_real16_T(&gpu_Squ_diff, Squ_diff);
    }
    Squared_Difference_GPU_kernel4<<<grid, block>>>(
        b_gpu_true_database, b_gpu_Meas_Cts_temp, c_true_database, i,
        Squ_diff_dim0, true_database_dim0, gpu_Squ_diff);
    CUDACHECK(hipGetLastError());
    Squ_diff_dirtyOnGpu = true;
  }
  emxFree_real16_T(&b_Meas_Cts_temp);
  emxFree_real16_T(&b_true_database);
  // sum_val=sum(diff,2);
  // tic;
  // [sort_val,sort_ind]=sort(gpuArray(sum_val));
  // [sort_val,sort_ind]=mink(sum_val,300);
  // toc;
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  if (Squ_diff_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_real16_T(Squ_diff, &gpu_Squ_diff);
  }
  gpuEmxFree_real32_T(&gpu_true_database);
  gpuEmxFree_real16_T(&b_gpu_true_database);
  gpuEmxFree_real_T(&gpu_Meas_Cts_temp);
  gpuEmxFree_real16_T(&b_gpu_Meas_Cts_temp);
  gpuEmxFree_real16_T(&gpu_Squ_diff);
#undef CUDACHECK
}

// End of code generation (Squared_Difference_GPU.cu)
