//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// Squared_Difference_GPU_terminate.cu
//
// Code generation for function 'Squared_Difference_GPU_terminate'
//

// Include files
#include "Squared_Difference_GPU_terminate.h"
#include "Squared_Difference_GPU_data.h"
#include "_coder_Squared_Difference_GPU_mex.h"
#include "rt_nonfinite.h"

// Function Definitions
void Squared_Difference_GPU_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void Squared_Difference_GPU_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (Squared_Difference_GPU_terminate.cu)
