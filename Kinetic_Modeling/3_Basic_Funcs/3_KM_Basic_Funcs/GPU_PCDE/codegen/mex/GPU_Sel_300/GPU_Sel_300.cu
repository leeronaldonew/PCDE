#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// GPU_Sel_300.cu
//
// Code generation for function 'GPU_Sel_300'
//

// Include files
#include "GPU_Sel_300.h"
#include "rt_nonfinite.h"
#include "rtwhalf.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include <cmath>

// Function Declarations
static __global__ void GPU_Sel_300_kernel1(const real32_T Gpu_Input_Bed_r[4352],
                                           const int32_T v,
                                           const real32_T true_db_sub_s[17000],
                                           real32_T a[17000]);

static __global__ void GPU_Sel_300_kernel10(const int32_T iwork[1000],
                                            const int32_T j, const int32_T kEnd,
                                            int32_T idx[1000]);

static __global__ void GPU_Sel_300_kernel11(const int32_T idx[1000],
                                            const int32_T q, const int32_T k,
                                            int32_T iwork[1000]);

static __global__ void GPU_Sel_300_kernel12(const int32_T idx[1000],
                                            const int32_T p, const int32_T k,
                                            int32_T iwork[1000]);

static __global__ void GPU_Sel_300_kernel13(const int32_T idx[1000],
                                            const int32_T p, const int32_T k,
                                            int32_T iwork[1000]);

static __global__ void GPU_Sel_300_kernel14(const int32_T idx[1000],
                                            const int32_T q, const int32_T k,
                                            int32_T iwork[1000]);

static __global__ void GPU_Sel_300_kernel2(const real32_T a[17000],
                                           real32_T y[17000]);

static __global__ void GPU_Sel_300_kernel3(const real32_T y[17000],
                                           real32_T sort_val_temp[1000]);

static __global__ void GPU_Sel_300_kernel4(const real32_T y[17000],
                                           const int32_T xoffset,
                                           real32_T sort_val_temp[1000]);

static __global__ void GPU_Sel_300_kernel5(const real32_T sort_val_temp[1000],
                                           int32_T idx[1000]);

static __global__ void GPU_Sel_300_kernel6(const real32_T sort_val_temp[1000],
                                           const int32_T idx[1000],
                                           real32_T ycol[1000]);

static __global__ void GPU_Sel_300_kernel7(const real32_T ycol[1000],
                                           real32_T sort_val_temp[1000]);

static __global__ void GPU_Sel_300_kernel8(const real32_T sort_val_temp[1000],
                                           const real_T s,
                                           const int32_T idx[1000],
                                           real32_T sort_sub[600]);

static __global__ void GPU_Sel_300_kernel9(const real32_T sort_sub[600],
                                           const int32_T v,
                                           __half params_s_sum_GPU[153600]);

// Function Definitions
static __global__ __launch_bounds__(512, 1) void GPU_Sel_300_kernel1(
    const real32_T Gpu_Input_Bed_r[4352], const int32_T v,
    const real32_T true_db_sub_s[17000], real32_T a[17000])
{
  uint64_T threadId;
  int32_T i;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId % 1000ULL);
  i = static_cast<int32_T>((threadId - static_cast<uint64_T>(k)) / 1000ULL);
  if ((static_cast<int32_T>(i < 17)) && (static_cast<int32_T>(k < 1000))) {
    a[k + 1000 * i] =
        true_db_sub_s[k + 1000 * i] - Gpu_Input_Bed_r[v + (i << 8)];
  }
}

static __global__ __launch_bounds__(1024, 1) void GPU_Sel_300_kernel10(
    const int32_T iwork[1000], const int32_T j, const int32_T kEnd,
    int32_T idx[1000])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(kEnd - 1);
  for (uint64_T b_idx{threadId}; b_idx <= loopEnd; b_idx += threadStride) {
    int32_T k;
    k = static_cast<int32_T>(b_idx);
    idx[(j + k) - 1] = iwork[k];
  }
}

static __global__ __launch_bounds__(32, 1) void GPU_Sel_300_kernel11(
    const int32_T idx[1000], const int32_T q, const int32_T k,
    int32_T iwork[1000])
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    iwork[k] = idx[q];
  }
}

static __global__ __launch_bounds__(32, 1) void GPU_Sel_300_kernel12(
    const int32_T idx[1000], const int32_T p, const int32_T k,
    int32_T iwork[1000])
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    iwork[k] = idx[p];
  }
}

static __global__ __launch_bounds__(32, 1) void GPU_Sel_300_kernel13(
    const int32_T idx[1000], const int32_T p, const int32_T k,
    int32_T iwork[1000])
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    iwork[k] = idx[p];
  }
}

static __global__ __launch_bounds__(32, 1) void GPU_Sel_300_kernel14(
    const int32_T idx[1000], const int32_T q, const int32_T k,
    int32_T iwork[1000])
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    iwork[k] = idx[q];
  }
}

static __global__
    __launch_bounds__(512, 1) void GPU_Sel_300_kernel2(const real32_T a[17000],
                                                       real32_T y[17000])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 17000) {
    real32_T f;
    f = a[k];
    y[k] = f * f;
  }
}

static __global__ __launch_bounds__(512, 1) void GPU_Sel_300_kernel3(
    const real32_T y[17000], real32_T sort_val_temp[1000])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 1000) {
    sort_val_temp[k] = y[k];
  }
}

static __global__ __launch_bounds__(512, 1) void GPU_Sel_300_kernel4(
    const real32_T y[17000], const int32_T xoffset,
    real32_T sort_val_temp[1000])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 1000) {
    sort_val_temp[k] += y[xoffset + k];
  }
}

static __global__ __launch_bounds__(512, 1) void GPU_Sel_300_kernel5(
    const real32_T sort_val_temp[1000], int32_T idx[1000])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 500) {
    real32_T f;
    real32_T f1;
    k = (k << 1) + 1;
    f = sort_val_temp[k - 1];
    f1 = sort_val_temp[k];
    if ((static_cast<int32_T>(
            (static_cast<int32_T>(
                (static_cast<int32_T>(f == f1)) ||
                (static_cast<int32_T>((static_cast<int32_T>(isnan(f))) &&
                                      (static_cast<int32_T>(isnan(f1))))))) ||
            (static_cast<int32_T>(f <= f1)))) ||
        (static_cast<int32_T>(isnan(f1)))) {
      idx[k - 1] = k;
      idx[k] = k + 1;
    } else {
      idx[k - 1] = k + 1;
      idx[k] = k;
    }
  }
}

static __global__ __launch_bounds__(512, 1) void GPU_Sel_300_kernel6(
    const real32_T sort_val_temp[1000], const int32_T idx[1000],
    real32_T ycol[1000])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 1000) {
    ycol[i] = sort_val_temp[idx[i] - 1];
  }
}

static __global__ __launch_bounds__(512, 1) void GPU_Sel_300_kernel7(
    const real32_T ycol[1000], real32_T sort_val_temp[1000])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 1000) {
    sort_val_temp[i] = ycol[i];
  }
}

static __global__ __launch_bounds__(320, 1) void GPU_Sel_300_kernel8(
    const real32_T sort_val_temp[1000], const real_T s, const int32_T idx[1000],
    real32_T sort_sub[600])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 300) {
    sort_sub[i] = static_cast<real32_T>(static_cast<real_T>(idx[i]) + s);
    sort_sub[i + 300] = sort_val_temp[i];
  }
}

static __global__ __launch_bounds__(512, 1) void GPU_Sel_300_kernel9(
    const real32_T sort_sub[600], const int32_T v,
    __half params_s_sum_GPU[153600])
{
  uint64_T threadId;
  int32_T i;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId % 2ULL);
  i = static_cast<int32_T>((threadId - static_cast<uint64_T>(k)) / 2ULL);
  if ((static_cast<int32_T>(i < 300)) && (static_cast<int32_T>(k < 2))) {
    params_s_sum_GPU[i + 300 * ((v << 1) + k)] =
        static_cast<__half>(sort_sub[i + 300 * k]);
  }
}

void GPU_Sel_300(const real32_T Gpu_Input_Bed_r[4352],
                 const real32_T true_db_sub_s[17000], real_T s,
                 real16_T params_s_sum_GPU[153600])
{
  dim3 block;
  dim3 grid;
  real_T b_s;
  int32_T idx[1000];
  int32_T(*gpu_idx)[1000];
  int32_T(*gpu_iwork)[1000];
  int32_T qEnd;
  real32_T(*gpu_a)[17000];
  real32_T(*gpu_true_db_sub_s)[17000];
  real32_T(*gpu_y)[17000];
  real32_T(*gpu_Gpu_Input_Bed_r)[4352];
  real32_T sort_val_temp[1000];
  real32_T(*gpu_sort_val_temp)[1000];
  real32_T(*gpu_ycol)[1000];
  real32_T(*gpu_sort_sub)[600];
  __half(*gpu_params_s_sum_GPU)[153600];
  boolean_T Gpu_Input_Bed_r_dirtyOnCpu;
  boolean_T params_s_sum_GPU_dirtyOnGpu;
  boolean_T true_db_sub_s_dirtyOnCpu;
  hipMalloc(&gpu_iwork, 4000ULL);
  hipMalloc(&gpu_params_s_sum_GPU, 307200ULL);
  hipMalloc(&gpu_sort_sub, 2400ULL);
  hipMalloc(&gpu_ycol, 4000ULL);
  hipMalloc(&gpu_idx, 4000ULL);
  hipMalloc(&gpu_sort_val_temp, 4000ULL);
  hipMalloc(&gpu_y, 68000ULL);
  hipMalloc(&gpu_a, 68000ULL);
  hipMalloc(&gpu_true_db_sub_s, 68000ULL);
  hipMalloc(&gpu_Gpu_Input_Bed_r, 17408ULL);
  params_s_sum_GPU_dirtyOnGpu = false;
  true_db_sub_s_dirtyOnCpu = true;
  Gpu_Input_Bed_r_dirtyOnCpu = true;
  //  # of voxels that you want to calculate simultaneously!
  b_s = (s - 1.0) * 1000.0;
  for (int32_T v{0}; v < 256; v++) {
    int32_T i;
    int32_T k;
    boolean_T idx_dirtyOnGpu;
    if (Gpu_Input_Bed_r_dirtyOnCpu) {
      hipMemcpy(*gpu_Gpu_Input_Bed_r, Gpu_Input_Bed_r, 17408ULL,
                 hipMemcpyHostToDevice);
    }
    Gpu_Input_Bed_r_dirtyOnCpu = false;
    if (true_db_sub_s_dirtyOnCpu) {
      hipMemcpy(*gpu_true_db_sub_s, true_db_sub_s, 68000ULL,
                 hipMemcpyHostToDevice);
    }
    true_db_sub_s_dirtyOnCpu = false;
    GPU_Sel_300_kernel1<<<dim3(34U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_Gpu_Input_Bed_r, v, *gpu_true_db_sub_s, *gpu_a);
    GPU_Sel_300_kernel2<<<dim3(34U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_a,
                                                                   *gpu_y);
    GPU_Sel_300_kernel3<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_y, *gpu_sort_val_temp);
    params_s_sum_GPU_dirtyOnGpu = true;
    for (k = 0; k < 16; k++) {
      GPU_Sel_300_kernel4<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
          *gpu_y, (k + 1) * 1000, *gpu_sort_val_temp);
    }
    GPU_Sel_300_kernel5<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_sort_val_temp, *gpu_idx);
    idx_dirtyOnGpu = true;
    i = 2;
    while (i < 1000) {
      int32_T i2;
      int32_T j;
      i2 = i << 1;
      j = 1;
      for (int32_T xoffset{i + 1}; xoffset < 1001; xoffset = qEnd + i) {
        int32_T kEnd;
        int32_T p;
        int32_T q;
        boolean_T validLaunchParams;
        p = j - 1;
        q = xoffset - 1;
        qEnd = j + i2;
        if (qEnd > 1001) {
          qEnd = 1001;
        }
        k = 0;
        kEnd = qEnd - j;
        while (k + 1 <= kEnd) {
          if (params_s_sum_GPU_dirtyOnGpu) {
            hipMemcpy(sort_val_temp, *gpu_sort_val_temp, 4000ULL,
                       hipMemcpyDeviceToHost);
          }
          params_s_sum_GPU_dirtyOnGpu = false;
          if (idx_dirtyOnGpu) {
            hipMemcpy(idx, *gpu_idx, 4000ULL, hipMemcpyDeviceToHost);
          }
          idx_dirtyOnGpu = false;
          if ((sort_val_temp[idx[p] - 1] == sort_val_temp[idx[q] - 1]) ||
              (std::isnan(sort_val_temp[idx[p] - 1]) &&
               std::isnan(sort_val_temp[idx[q] - 1])) ||
              (sort_val_temp[idx[p] - 1] <= sort_val_temp[idx[q] - 1]) ||
              std::isnan(sort_val_temp[idx[q] - 1])) {
            GPU_Sel_300_kernel13<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
                *gpu_idx, p, k, *gpu_iwork);
            p++;
            if (p + 1 == xoffset) {
              while (q + 1 < qEnd) {
                k++;
                GPU_Sel_300_kernel14<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
                    *gpu_idx, q, k, *gpu_iwork);
                q++;
              }
            }
          } else {
            GPU_Sel_300_kernel11<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
                *gpu_idx, q, k, *gpu_iwork);
            q++;
            if (q + 1 == qEnd) {
              while (p + 1 < xoffset) {
                k++;
                GPU_Sel_300_kernel12<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
                    *gpu_idx, p, k, *gpu_iwork);
                p++;
              }
            }
          }
          k++;
        }
        validLaunchParams =
            mwGetLaunchParameters1D(static_cast<real_T>((kEnd - 1) + 1LL),
                                    &grid, &block, 1024U, 65535U);
        if (validLaunchParams) {
          GPU_Sel_300_kernel10<<<grid, block>>>(*gpu_iwork, j, kEnd, *gpu_idx);
          idx_dirtyOnGpu = true;
        }
        j = qEnd;
      }
      i = i2;
    }
    GPU_Sel_300_kernel6<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_sort_val_temp, *gpu_idx, *gpu_ycol);
    GPU_Sel_300_kernel7<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_ycol, *gpu_sort_val_temp);
    GPU_Sel_300_kernel8<<<dim3(1U, 1U, 1U), dim3(320U, 1U, 1U)>>>(
        *gpu_sort_val_temp, b_s, *gpu_idx, *gpu_sort_sub);
    GPU_Sel_300_kernel9<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_sort_sub, v, *gpu_params_s_sum_GPU);
    params_s_sum_GPU_dirtyOnGpu = true;
  }
  if (params_s_sum_GPU_dirtyOnGpu) {
    hipMemcpy(params_s_sum_GPU, *gpu_params_s_sum_GPU, 307200ULL,
               hipMemcpyDeviceToHost);
  }
  hipFree(*gpu_Gpu_Input_Bed_r);
  hipFree(*gpu_true_db_sub_s);
  hipFree(*gpu_a);
  hipFree(*gpu_y);
  hipFree(*gpu_sort_val_temp);
  hipFree(*gpu_idx);
  hipFree(*gpu_ycol);
  hipFree(*gpu_sort_sub);
  hipFree(*gpu_params_s_sum_GPU);
  hipFree(*gpu_iwork);
}

// End of code generation (GPU_Sel_300.cu)
