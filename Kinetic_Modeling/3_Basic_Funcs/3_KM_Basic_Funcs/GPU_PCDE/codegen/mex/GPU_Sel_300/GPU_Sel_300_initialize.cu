//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// GPU_Sel_300_initialize.cu
//
// Code generation for function 'GPU_Sel_300_initialize'
//

// Include files
#include "GPU_Sel_300_initialize.h"
#include "GPU_Sel_300_data.h"
#include "_coder_GPU_Sel_300_mex.h"
#include "rt_nonfinite.h"

// Function Definitions
void GPU_Sel_300_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal,
                          (const char_T *)"distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal,
                          (const char_T *)"matlab_coder or fixed_point_toolbox",
                          1);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (GPU_Sel_300_initialize.cu)
