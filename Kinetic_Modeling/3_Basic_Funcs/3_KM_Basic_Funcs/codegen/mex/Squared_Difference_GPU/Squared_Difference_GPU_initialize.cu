//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// Squared_Difference_GPU_initialize.cu
//
// Code generation for function 'Squared_Difference_GPU_initialize'
//

// Include files
#include "Squared_Difference_GPU_initialize.h"
#include "Squared_Difference_GPU_data.h"
#include "_coder_Squared_Difference_GPU_mex.h"
#include "rt_nonfinite.h"

// Function Definitions
void Squared_Difference_GPU_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal,
                          (const char_T *)"distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal,
                          (const char_T *)"matlab_coder or fixed_point_toolbox",
                          1);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (Squared_Difference_GPU_initialize.cu)
