//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// TTCM_analytic_Multi_GPU_terminate.cu
//
// Code generation for function 'TTCM_analytic_Multi_GPU_terminate'
//

// Include files
#include "TTCM_analytic_Multi_GPU_terminate.h"
#include "TTCM_analytic_Multi_GPU_data.h"
#include "_coder_TTCM_analytic_Multi_GPU_mex.h"
#include "rt_nonfinite.h"

// Function Definitions
void TTCM_analytic_Multi_GPU_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void TTCM_analytic_Multi_GPU_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (TTCM_analytic_Multi_GPU_terminate.cu)
