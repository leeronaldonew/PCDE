//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// TTCM_analytic_Multi_GPU_initialize.cu
//
// Code generation for function 'TTCM_analytic_Multi_GPU_initialize'
//

// Include files
#include "TTCM_analytic_Multi_GPU_initialize.h"
#include "TTCM_analytic_Multi_GPU_data.h"
#include "_coder_TTCM_analytic_Multi_GPU_mex.h"
#include "rt_nonfinite.h"

// Function Definitions
void TTCM_analytic_Multi_GPU_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal,
                          (const char_T *)"distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal,
                          (const char_T *)"matlab_coder or fixed_point_toolbox",
                          1);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (TTCM_analytic_Multi_GPU_initialize.cu)
