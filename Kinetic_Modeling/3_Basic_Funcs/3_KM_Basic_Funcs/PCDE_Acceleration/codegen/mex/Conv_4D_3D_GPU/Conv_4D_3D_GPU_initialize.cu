//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// Conv_4D_3D_GPU_initialize.cu
//
// Code generation for function 'Conv_4D_3D_GPU_initialize'
//

// Include files
#include "Conv_4D_3D_GPU_initialize.h"
#include "Conv_4D_3D_GPU_data.h"
#include "_coder_Conv_4D_3D_GPU_mex.h"
#include "rt_nonfinite.h"

// Function Definitions
void Conv_4D_3D_GPU_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal,
                          (const char_T *)"distrib_computing_toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (Conv_4D_3D_GPU_initialize.cu)
