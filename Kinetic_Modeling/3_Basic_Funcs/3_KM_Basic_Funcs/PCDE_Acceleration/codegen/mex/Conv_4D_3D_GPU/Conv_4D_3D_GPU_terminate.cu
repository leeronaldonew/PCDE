//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// Conv_4D_3D_GPU_terminate.cu
//
// Code generation for function 'Conv_4D_3D_GPU_terminate'
//

// Include files
#include "Conv_4D_3D_GPU_terminate.h"
#include "Conv_4D_3D_GPU_data.h"
#include "_coder_Conv_4D_3D_GPU_mex.h"
#include "rt_nonfinite.h"

// Function Definitions
void Conv_4D_3D_GPU_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void Conv_4D_3D_GPU_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (Conv_4D_3D_GPU_terminate.cu)
