#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// Conv_4D_3D_GPU.cu
//
// Code generation for function 'Conv_4D_3D_GPU'
//

// Include files
#include "Conv_4D_3D_GPU.h"
#include "Conv_4D_3D_GPU_emxutil.h"
#include "Conv_4D_3D_GPU_types.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include <cstring>

// Function Declarations
static __global__ void Conv_4D_3D_GPU_kernel1(const int32_T size_WB,
                                              emxArray_real32_T DB_3D);

static void gpuEmxFree_real32_T(emxArray_real32_T *gpu);

static void gpuEmxMemcpyCpuToGpu_real32_T(emxArray_real32_T *gpu,
                                          const emxArray_real32_T *cpu);

static void gpuEmxMemcpyGpuToCpu_real32_T(emxArray_real32_T *cpu,
                                          emxArray_real32_T *gpu);

static void gpuEmxReset_real32_T(emxArray_real32_T *gpu);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void Conv_4D_3D_GPU_kernel1(
    const int32_T size_WB, emxArray_real32_T DB_3D)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(size_WB);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    DB_3D.data[i] = 0.0F;
  }
}

static void gpuEmxFree_real32_T(emxArray_real32_T *gpu)
{
  if (gpu->data != (void *)4207599121ULL) {
    hipFree(gpu->data);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxMemcpyCpuToGpu_real32_T(emxArray_real32_T *gpu,
                                          const emxArray_real32_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  if (gpu->numDimensions < cpu->numDimensions) {
    gpu->numDimensions = cpu->numDimensions;
    emlrtFreeMex(gpu->size);
    gpu->size = (int32_T *)emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T));
  } else {
    gpu->numDimensions = cpu->numDimensions;
  }
  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < actualSize) {
    if (gpu->canFreeData) {
      hipFree(gpu->data);
    }
    i = cpu->allocatedSize;
    if (i < actualSize) {
      i = actualSize;
    }
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
    hipMalloc(&gpu->data, gpu->allocatedSize * sizeof(real32_T));
  }
  hipMemcpy(gpu->data, cpu->data, actualSize * sizeof(real32_T),
             hipMemcpyHostToDevice);
}

static void gpuEmxMemcpyGpuToCpu_real32_T(emxArray_real32_T *cpu,
                                          emxArray_real32_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  hipMemcpy(cpu->data, gpu->data, actualSize * sizeof(real32_T),
             hipMemcpyDeviceToHost);
}

static void gpuEmxReset_real32_T(emxArray_real32_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_real32_T));
}

void Conv_4D_3D_GPU(const emxArray_real32_T *kBq, emxArray_real32_T *DB_3D)
{
  dim3 block;
  dim3 grid;
  emxArray_real32_T gpu_DB_3D;
  int32_T b_size_WB;
  int32_T i;
  uint32_T size_WB[4];
  boolean_T DB_3D_dirtyOnGpu;
  boolean_T validLaunchParams;
  gpuEmxReset_real32_T(&gpu_DB_3D);
  DB_3D_dirtyOnGpu = false;
  //  4D array into 3D array
  for (i = 0; i < 4; i++) {
    size_WB[i] = static_cast<uint32_T>(kBq->size[i]);
  }
  i = DB_3D->size[0] * DB_3D->size[1] * DB_3D->size[2];
  DB_3D->size[0] = static_cast<int32_T>(static_cast<real_T>(size_WB[0]) *
                                        static_cast<real_T>(size_WB[1]));
  DB_3D->size[1] = static_cast<int32_T>(size_WB[3]);
  DB_3D->size[2] = static_cast<int32_T>(size_WB[2]);
  emxEnsureCapacity_real32_T(DB_3D, i);
  b_size_WB = static_cast<int32_T>(static_cast<real_T>(size_WB[0]) *
                                   static_cast<real_T>(size_WB[1])) *
                  static_cast<int32_T>(size_WB[3]) *
                  static_cast<int32_T>(size_WB[2]) -
              1;
  validLaunchParams = mwGetLaunchParameters1D(
      static_cast<real_T>(b_size_WB + 1LL), &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    gpuEmxMemcpyCpuToGpu_real32_T(&gpu_DB_3D, DB_3D);
    Conv_4D_3D_GPU_kernel1<<<grid, block>>>(b_size_WB, gpu_DB_3D);
    DB_3D_dirtyOnGpu = true;
  }
  i = static_cast<int32_T>(size_WB[2]);
  for (int32_T k{0}; k < i; k++) {
    b_size_WB = static_cast<int32_T>(size_WB[1]);
    for (int32_T j{0}; j < b_size_WB; j++) {
      int32_T i2;
      i2 = static_cast<int32_T>(size_WB[0]);
      for (int32_T b_i{0}; b_i < i2; b_i++) {
        int32_T i3;
        i3 = static_cast<int32_T>(size_WB[3]);
        for (int32_T p{0}; p < i3; p++) {
          if (DB_3D_dirtyOnGpu) {
            gpuEmxMemcpyGpuToCpu_real32_T(DB_3D, &gpu_DB_3D);
          }
          DB_3D->data[((static_cast<int32_T>(
                            (static_cast<real_T>(b_i) + 1.0) +
                            static_cast<real_T>(size_WB[0]) *
                                ((static_cast<real_T>(j) + 1.0) - 1.0)) +
                        DB_3D->size[0] * p) +
                       DB_3D->size[0] * DB_3D->size[1] * k) -
                      1] =
              kBq->data[((b_i + kBq->size[0] * j) +
                         kBq->size[0] * kBq->size[1] * k) +
                        kBq->size[0] * kBq->size[1] * kBq->size[2] * p];
          DB_3D_dirtyOnGpu = false;
        }
      }
    }
  }
  if (DB_3D_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_real32_T(DB_3D, &gpu_DB_3D);
  }
  gpuEmxFree_real32_T(&gpu_DB_3D);
}

// End of code generation (Conv_4D_3D_GPU.cu)
