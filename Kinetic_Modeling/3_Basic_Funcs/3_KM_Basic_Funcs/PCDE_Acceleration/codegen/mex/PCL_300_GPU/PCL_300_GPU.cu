#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// PCL_300_GPU.cu
//
// Code generation for function 'PCL_300_GPU'
//

// Include files
#include "PCL_300_GPU.h"
#include "PCL_300_GPU_types.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWMemoryManager.hpp"
#include "MWShuffleUtility.h"
#include "MWSortFunctors.h"
#include "MWSortWithIndexUtilityHost.h"

// Function Declarations
static __global__ void PCL_300_GPU_kernel1(const real32_T Slice_TACs_data[],
                                           const int32_T Slice_TACs_size[2],
                                           const int32_T v, const int32_T i,
                                           boolean_T x_data[17]);

static __global__ void PCL_300_GPU_kernel10(const int32_T v,
                                            real32_T Ind_300_data[]);

static __global__ void PCL_300_GPU_kernel2(const real32_T Slice_TACs_data[],
                                           const int32_T Slice_TACs_size[2],
                                           const int32_T v,
                                           const real32_T DB_data[],
                                           const int32_T DB_size,
                                           real32_T a_data[8500000]);

static __global__ void PCL_300_GPU_kernel3(const real32_T a_data[8500000],
                                           const int32_T nx,
                                           real32_T y_data[8500000]);

static __global__ void PCL_300_GPU_kernel4(const real32_T y_data[8500000],
                                           real32_T out[500000]);

static __global__ void PCL_300_GPU_kernel5(const real32_T y_data[8500000],
                                           const int32_T xoffset,
                                           real32_T out[500000]);

static __global__ void PCL_300_GPU_kernel6(real32_T out[500000]);

static __global__ void PCL_300_GPU_kernel7(int32_T inDims[2]);

static __global__ void PCL_300_GPU_kernel8(real_T idx[500000]);

static __global__ void PCL_300_GPU_kernel9(const real_T idx[500000],
                                           const int32_T v,
                                           real32_T Ind_300_data[]);

static void binary_expand_op(real32_T a_data[], int32_T a_size[2],
                             const real32_T DB_data[], const int32_T DB_size[2],
                             const real32_T Slice_TACs_data[],
                             const int32_T Slice_TACs_size[2], int32_T v);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void PCL_300_GPU_kernel1(
    const real32_T Slice_TACs_data[], const int32_T Slice_TACs_size[2],
    const int32_T v, const int32_T i, boolean_T x_data[17])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(i);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    x_data[k] = (Slice_TACs_data[v + Slice_TACs_size[0] * k] == 0.0F);
  }
}

static __global__
    __launch_bounds__(320, 1) void PCL_300_GPU_kernel10(const int32_T v,
                                                        real32_T Ind_300_data[])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 300) {
    Ind_300_data[k + 300 * v] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void PCL_300_GPU_kernel2(
    const real32_T Slice_TACs_data[], const int32_T Slice_TACs_size[2],
    const int32_T v, const real32_T DB_data[], const int32_T DB_size,
    real32_T a_data[8500000])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = 500000ULL * (static_cast<uint64_T>(DB_size) + 1ULL) - 1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i1;
    int32_T k;
    i1 = static_cast<int32_T>(idx % 500000ULL);
    k = static_cast<int32_T>((idx - static_cast<uint64_T>(i1)) / 500000ULL);
    a_data[i1 + 500000 * k] =
        DB_data[i1 + 500000 * k] - Slice_TACs_data[v + Slice_TACs_size[0] * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void PCL_300_GPU_kernel3(
    const real32_T a_data[8500000], const int32_T nx, real32_T y_data[8500000])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(nx - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    real32_T f;
    k = static_cast<int32_T>(idx);
    f = a_data[k];
    y_data[k] = f * f;
  }
}

static __global__ __launch_bounds__(512, 1) void PCL_300_GPU_kernel4(
    const real32_T y_data[8500000], real32_T out[500000])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 500000) {
    out[k] = y_data[k];
  }
}

static __global__ __launch_bounds__(512, 1) void PCL_300_GPU_kernel5(
    const real32_T y_data[8500000], const int32_T xoffset, real32_T out[500000])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 500000) {
    out[k] += y_data[xoffset + k];
  }
}

static __global__
    __launch_bounds__(512, 1) void PCL_300_GPU_kernel6(real32_T out[500000])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 500000) {
    out[k] = 0.0F;
  }
}

static __global__
    __launch_bounds__(32, 1) void PCL_300_GPU_kernel7(int32_T inDims[2])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 2) {
    inDims[k] = -499999 * k + 500000;
  }
}

static __global__
    __launch_bounds__(512, 1) void PCL_300_GPU_kernel8(real_T idx[500000])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 500000) {
    idx[k] = 0.0;
  }
}

static __global__ __launch_bounds__(320, 1) void PCL_300_GPU_kernel9(
    const real_T idx[500000], const int32_T v, real32_T Ind_300_data[])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 300) {
    Ind_300_data[k + 300 * v] = static_cast<real32_T>(idx[k]);
  }
}

static void binary_expand_op(real32_T a_data[], int32_T a_size[2],
                             const real32_T DB_data[], const int32_T DB_size[2],
                             const real32_T Slice_TACs_data[],
                             const int32_T Slice_TACs_size[2], int32_T v)
{
  int32_T aux_0_1;
  int32_T aux_1_1;
  int32_T i;
  int32_T stride_0_1;
  int32_T stride_1_1;
  i = Slice_TACs_size[1];
  a_size[0] = 500000;
  if (i == 1) {
    a_size[1] = DB_size[1];
  } else {
    a_size[1] = i;
  }
  stride_0_1 = (DB_size[1] != 1);
  stride_1_1 = (i != 1);
  aux_0_1 = 0;
  aux_1_1 = 0;
  if (i == 1) {
    i = DB_size[1];
  }
  for (int32_T i1{0}; i1 < i; i1++) {
    for (int32_T i2{0}; i2 < 500000; i2++) {
      a_data[i2 + 500000 * i1] =
          DB_data[i2 + 500000 * aux_0_1] -
          Slice_TACs_data[v + Slice_TACs_size[0] * aux_1_1];
    }
    aux_1_1 += stride_1_1;
    aux_0_1 += stride_0_1;
  }
}

void PCL_300_GPU(PCL_300_GPUStackData *SD, const real32_T DB_data[],
                 const int32_T DB_size[2], const real32_T Slice_TACs_data[],
                 const int32_T Slice_TACs_size[2], real32_T Ind_300_data[],
                 int32_T Ind_300_size[2])
{
  dim3 block;
  dim3 grid;
  real_T(*gpu_idx)[500000];
  int32_T a_size[2];
  int32_T inDims[2];
  int32_T x_size[2];
  int32_T(*gpu_Slice_TACs_size)[2];
  int32_T(*gpu_inDims)[2];
  int32_T b_i;
  int32_T i;
  real32_T(*gpu_a_data)[8500000];
  real32_T(*gpu_y_data)[8500000];
  real32_T(*gpu_out)[500000];
  real32_T *gpu_DB_data;
  real32_T *gpu_Ind_300_data;
  real32_T *gpu_Slice_TACs_data;
  boolean_T x_data[17];
  boolean_T(*gpu_x_data)[17];
  boolean_T DB_data_dirtyOnCpu;
  boolean_T Ind_300_data_dirtyOnGpu;
  boolean_T Slice_TACs_data_dirtyOnCpu;
  boolean_T Slice_TACs_size_dirtyOnCpu;
  boolean_T a_data_dirtyOnCpu;
  boolean_T a_data_dirtyOnGpu;
  boolean_T x_data_dirtyOnGpu;
  mwCudaMalloc(&gpu_idx, 4000000ULL);
  mwCudaMalloc(&gpu_inDims, 8ULL);
  mwCudaMalloc(&gpu_out, 2000000ULL);
  mwCudaMalloc(&gpu_y_data, 34000000ULL);
  mwCudaMalloc(&gpu_a_data, 34000000ULL);
  mwCudaMalloc(&gpu_DB_data,
               static_cast<uint64_T>(8500000U * sizeof(real32_T)));
  mwCudaMalloc(&gpu_Ind_300_data,
               static_cast<uint64_T>(19660800U * sizeof(real32_T)));
  mwCudaMalloc(&gpu_x_data, 17ULL);
  mwCudaMalloc(&gpu_Slice_TACs_size, 8ULL);
  mwCudaMalloc(&gpu_Slice_TACs_data,
               static_cast<uint64_T>(1114112U * sizeof(real32_T)));
  a_data_dirtyOnGpu = false;
  Ind_300_data_dirtyOnGpu = false;
  x_data_dirtyOnGpu = false;
  a_data_dirtyOnCpu = false;
  DB_data_dirtyOnCpu = true;
  Slice_TACs_size_dirtyOnCpu = true;
  Slice_TACs_data_dirtyOnCpu = true;
  // PI_Time=[10:5:90];
  // [permutes,true_data]=make_database_NH_FDG_Full(PI_Time); % k1:0.01~1, k2:
  // 0.01~1, k3:0.01~0.5, k4=0
  //  Seclting a 300 Parameter Combination List
  // Num_Vox=size(Slice_TACs,1);
  // Num_Time=size(Slice_Time,2);
  // Num_Comb=size(DB,1);
  // tic;
  // [sort_val,sort_ind]= mink(sum( (DB-Voxel_TAC).^(2),2 ),300);
  // toc;
  Ind_300_size[0] = 300;
  Ind_300_size[1] = Slice_TACs_size[0];
  i = Slice_TACs_size[0];
  if (0 <= Slice_TACs_size[0] - 1) {
    b_i = Slice_TACs_size[1] - 1;
    x_size[1] = Slice_TACs_size[1];
  }
  for (int32_T v{0}; v < i; v++) {
    int32_T k;
    Ind_300_data_dirtyOnGpu = mwGetLaunchParameters1D(
        static_cast<real_T>(b_i + 1LL), &grid, &block, 1024U, 65535U);
    if (Ind_300_data_dirtyOnGpu) {
      if (Slice_TACs_data_dirtyOnCpu) {
        hipMemcpy(gpu_Slice_TACs_data, Slice_TACs_data,
                   Slice_TACs_size[0] * Slice_TACs_size[1] * sizeof(real32_T),
                   hipMemcpyHostToDevice);
      }
      Slice_TACs_data_dirtyOnCpu = false;
      if (Slice_TACs_size_dirtyOnCpu) {
        hipMemcpy(*gpu_Slice_TACs_size, Slice_TACs_size, 8ULL,
                   hipMemcpyHostToDevice);
      }
      Slice_TACs_size_dirtyOnCpu = false;
      PCL_300_GPU_kernel1<<<grid, block>>>(
          gpu_Slice_TACs_data, *gpu_Slice_TACs_size, v, b_i, *gpu_x_data);
      x_data_dirtyOnGpu = true;
    }
    Ind_300_data_dirtyOnGpu = (x_size[1] != 0);
    if (Ind_300_data_dirtyOnGpu) {
      boolean_T exitg1;
      k = 1;
      exitg1 = false;
      while ((!exitg1) && (k <= x_size[1])) {
        if (x_data_dirtyOnGpu) {
          hipMemcpy(x_data, *gpu_x_data, 17ULL, hipMemcpyDeviceToHost);
        }
        x_data_dirtyOnGpu = false;
        if (!x_data[k - 1]) {
          Ind_300_data_dirtyOnGpu = false;
          exitg1 = true;
        } else {
          k++;
        }
      }
    }
    if (Ind_300_data_dirtyOnGpu) {
      PCL_300_GPU_kernel10<<<dim3(1U, 1U, 1U), dim3(320U, 1U, 1U)>>>(
          v, gpu_Ind_300_data);
      Ind_300_data_dirtyOnGpu = true;
    } else {
      int32_T nx;
      if (DB_size[1] == Slice_TACs_size[1]) {
        a_size[0] = 500000;
        a_size[1] = DB_size[1];
        Ind_300_data_dirtyOnGpu = mwGetLaunchParameters1D(
            static_cast<real_T>(500000LL * ((DB_size[1] - 1) + 1LL)), &grid,
            &block, 1024U, 65535U);
        if (Ind_300_data_dirtyOnGpu) {
          if (Slice_TACs_data_dirtyOnCpu) {
            hipMemcpy(gpu_Slice_TACs_data, Slice_TACs_data,
                       Slice_TACs_size[0] * Slice_TACs_size[1] *
                           sizeof(real32_T),
                       hipMemcpyHostToDevice);
          }
          Slice_TACs_data_dirtyOnCpu = false;
          if (Slice_TACs_size_dirtyOnCpu) {
            hipMemcpy(*gpu_Slice_TACs_size, Slice_TACs_size, 8ULL,
                       hipMemcpyHostToDevice);
          }
          Slice_TACs_size_dirtyOnCpu = false;
          if (DB_data_dirtyOnCpu) {
            hipMemcpy(gpu_DB_data, DB_data,
                       500000 * DB_size[1] * sizeof(real32_T),
                       hipMemcpyHostToDevice);
          }
          DB_data_dirtyOnCpu = false;
          if (a_data_dirtyOnCpu) {
            hipMemcpy(*gpu_a_data, SD->f0.a_data, 34000000ULL,
                       hipMemcpyHostToDevice);
          }
          PCL_300_GPU_kernel2<<<grid, block>>>(
              gpu_Slice_TACs_data, *gpu_Slice_TACs_size, v, gpu_DB_data,
              DB_size[1] - 1, *gpu_a_data);
          a_data_dirtyOnCpu = false;
          a_data_dirtyOnGpu = true;
        }
      } else {
        if (a_data_dirtyOnGpu) {
          hipMemcpy(SD->f0.a_data, *gpu_a_data, 34000000ULL,
                     hipMemcpyDeviceToHost);
        }
        binary_expand_op(SD->f0.a_data, a_size, DB_data, DB_size,
                         Slice_TACs_data, Slice_TACs_size, v);
        a_data_dirtyOnGpu = false;
        a_data_dirtyOnCpu = true;
      }
      nx = 500000 * a_size[1];
      Ind_300_data_dirtyOnGpu = mwGetLaunchParameters1D(
          static_cast<real_T>((nx - 1) + 1LL), &grid, &block, 1024U, 65535U);
      if (Ind_300_data_dirtyOnGpu) {
        if (a_data_dirtyOnCpu) {
          hipMemcpy(*gpu_a_data, SD->f0.a_data, 34000000ULL,
                     hipMemcpyHostToDevice);
        }
        a_data_dirtyOnCpu = false;
        PCL_300_GPU_kernel3<<<grid, block>>>(*gpu_a_data, nx, *gpu_y_data);
      }
      nx = a_size[1];
      if (a_size[1] == 0) {
        PCL_300_GPU_kernel6<<<dim3(977U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
            *gpu_out);
      } else {
        PCL_300_GPU_kernel4<<<dim3(977U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
            *gpu_y_data, *gpu_out);
        for (k = 0; k <= nx - 2; k++) {
          PCL_300_GPU_kernel5<<<dim3(977U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
              *gpu_y_data, (k + 1) * 500000, *gpu_out);
        }
      }
      PCL_300_GPU_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_inDims);
      PCL_300_GPU_kernel8<<<dim3(977U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_idx);
      hipMemcpy(inDims, *gpu_inDims, 8ULL, hipMemcpyDeviceToHost);
      thrustSortImplWithIndex(&(*gpu_out)[0], &(*gpu_idx)[0], 2, &inDims[0], 1,
                              'a', false);
      PCL_300_GPU_kernel9<<<dim3(1U, 1U, 1U), dim3(320U, 1U, 1U)>>>(
          *gpu_idx, v, gpu_Ind_300_data);
      Ind_300_data_dirtyOnGpu = true;
    }
  }
  if (Ind_300_data_dirtyOnGpu) {
    hipMemcpy(Ind_300_data, gpu_Ind_300_data,
               300 * Ind_300_size[1] * sizeof(real32_T),
               hipMemcpyDeviceToHost);
  }
  mwCudaFree(&gpu_Slice_TACs_data[0]);
  mwCudaFree(&(*gpu_Slice_TACs_size)[0]);
  mwCudaFree(&(*gpu_x_data)[0]);
  mwCudaFree(&gpu_Ind_300_data[0]);
  mwCudaFree(&gpu_DB_data[0]);
  mwCudaFree(&(*gpu_a_data)[0]);
  mwCudaFree(&(*gpu_y_data)[0]);
  mwCudaFree(&(*gpu_out)[0]);
  mwCudaFree(&(*gpu_inDims)[0]);
  mwCudaFree(&(*gpu_idx)[0]);
}

// End of code generation (PCL_300_GPU.cu)
