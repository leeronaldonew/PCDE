//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// PCL_300_GPU_terminate.cu
//
// Code generation for function 'PCL_300_GPU_terminate'
//

// Include files
#include "PCL_300_GPU_terminate.h"
#include "PCL_300_GPU_data.h"
#include "_coder_PCL_300_GPU_mex.h"
#include "rt_nonfinite.h"

// Function Definitions
void PCL_300_GPU_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void PCL_300_GPU_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (PCL_300_GPU_terminate.cu)
